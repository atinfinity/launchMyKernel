#include "hip/hip_runtime.h"
#include "myKernel.cuh"

#include <opencv2/cudev.hpp>
#include <hip/hip_runtime.h>
#include <>

// ��f�l���l�K�|�W���]�����邾����CUDA�J�[�l��
__global__ void myKernel(const cv::cudev::GlobPtrSz<uchar> src, cv::cudev::GlobPtrSz<uchar> dst)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	dst.data[y*src.step + x] =  UCHAR_MAX - src.data[y*src.step + x];
}

void launchMyKernel(cv::cuda::GpuMat &src, cv::cuda::GpuMat &dst)
{
	cv::cudev::GlobPtrSz<uchar> pSrc = 
		cv::cudev::globPtr(src.ptr<uchar>(), src.step, src.rows, src.cols * src.channels());

	cv::cudev::GlobPtrSz<uchar> pDst = 
		cv::cudev::globPtr(dst.ptr<uchar>(), dst.step, dst.rows, dst.cols * dst.channels());

	const dim3 block(32, 8);
	const dim3 grid(cv::cudev::divUp(src.cols, block.x), cv::cudev::divUp(src.rows, block.y));

	// ����CUDA�J�[�l�����Ăяo��
	myKernel<<<grid, block>>>(pSrc, pDst);

	CV_CUDEV_SAFE_CALL(hipGetLastError());
	CV_CUDEV_SAFE_CALL(hipDeviceSynchronize());
}
