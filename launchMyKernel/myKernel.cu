#include "hip/hip_runtime.h"
#include "myKernel.cuh"

#include <opencv2/cudev.hpp>
#include <hip/hip_runtime.h>
#include <>

// 画素値をネガポジ反転させるだけのCUDAカーネル
__global__ void myKernel(const cv::cudev::GlobPtrSz<uchar> src, cv::cudev::GlobPtrSz<uchar> dst)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
    if((x < dst.cols) && (y < dst.rows)){
        dst.data[y*src.step + x] = UCHAR_MAX - src.data[y*src.step + x];
    }
}

void launchMyKernel(cv::cuda::GpuMat &src, cv::cuda::GpuMat &dst)
{
    const dim3 block(32, 8);
    const dim3 grid(cv::cudev::divUp(src.cols, block.x), cv::cudev::divUp(src.rows, block.y));

    // 自作CUDAカーネルを呼び出す
    myKernel<<<grid, block>>>(src, dst);

    CV_CUDEV_SAFE_CALL(hipGetLastError());
    CV_CUDEV_SAFE_CALL(hipDeviceSynchronize());
}
